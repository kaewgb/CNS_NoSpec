#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "header.h"
#include "helper_functions.h"
#define CUDA_SAFE_CALL( call )                               						\
{                                                              						\
    hipError_t err = call;                                                       	\
    if( hipSuccess != err) {                                                    	\
		fprintf(stderr, "Cuda error in call at file '%s' in line %i : %s.\n",     	\
				__FILE__, __LINE__, hipGetErrorString( err) );             		\
		exit(-1);                                                             		\
	}                                         										\
}

extern global_const_t h_const;
extern global_const_t *d_const_ptr;

void gpu_allocate_3D(double *&d_ptr, int dim[]){
	CUDA_SAFE_CALL(hipMalloc((void **) &d_ptr, dim[0]*dim[1]*dim[2] * sizeof(double)));
}
void gpu_copy_to_host_3D(double ***host, double *dev, int dim[]){
	CUDA_SAFE_CALL(hipMemcpy(host[0][0], dev, dim[0]*dim[1]*dim[2] * sizeof(double), hipMemcpyDeviceToHost));
}
void gpu_free_3D(double *d_ptr){
	CUDA_SAFE_CALL(hipFree(d_ptr));
}

void gpu_allocate_4D(double *&d_ptr, int dim[], int dl){
	CUDA_SAFE_CALL(hipMalloc((void **) &d_ptr, dim[0]*dim[1]*dim[2]*dl * sizeof(double)));
}

void gpu_copy_from_host_4D(double *dev, double ****host, int dim[], int dl){
	CUDA_SAFE_CALL(hipMemcpy(dev, host[0][0][0], dim[0]*dim[1]*dim[2]*dl * sizeof(double), hipMemcpyHostToDevice));
}

void gpu_copy_to_host_4D(double ****host, double *dev, int dim[], int dl){
	CUDA_SAFE_CALL(hipMemcpy(host[0][0][0], dev, dim[0]*dim[1]*dim[2]*dl * sizeof(double), hipMemcpyDeviceToHost));
}

void gpu_free_4D(double *d_ptr){
	CUDA_SAFE_CALL(hipFree(d_ptr));
}

#define d_ptr(l,i,j,k)	d_ptr[(l)*g->comp_offset_g + (i)*g->plane_offset_g + (j)*g->dim_g[2] + (k)]

__device__ kernel_const_t k_const;
__global__ void gpu_fill_boundary_z_kernel(
	global_const_t *g, 		// i: Global Constants
	double *d_ptr			// i/o:	Device Pointer
){
	int i,j,k,l;
	i = blockIdx.x * blockDim.x + threadIdx.x;
	j = blockIdx.y * blockDim.y + threadIdx.y;

	if(i < g->dim[0] && j < g->dim[1]){
		i += g->ng;
		j += g->ng;
		FOR(l, 0, g->nc){
			FOR(k, 0, g->ng){
				d_ptr(l,i,j,k) 					= d_ptr(l,i,j,k+g->dim[2]);
				d_ptr(l,i,j,k+g->dim[2]+g->ng)	= d_ptr(l,i,j,k+g->ng);
			}
		}
	}

}
__global__ void gpu_fill_boundary_y_kernel(
	global_const_t *g, 		// i: Global Constants
	double *d_ptr			// i/o:	Device Pointer
){
	int i,j,k,l;
	i = blockIdx.x * blockDim.x + threadIdx.x;
	k = blockIdx.y * blockDim.y + threadIdx.y;

	if(i < g->dim[0] && k < g->dim_g[2]){
		i += g->ng;
		FOR(l, 0, g->nc){
			FOR(j, 0, g->ng){
				d_ptr(l,i,j,k) 					= d_ptr(l,i,j+g->dim[1],k);
				d_ptr(l,i,j+g->dim[1]+g->ng,k)	= d_ptr(l,i,j+g->ng,k);
			}
		}
	}

}
__global__ void gpu_fill_boundary_x_kernel(
	global_const_t *g, 		// i: Global Constants
	double *d_ptr			// i/o:	Device Pointer
){
	int i,j,k,l;
	j = blockIdx.x * blockDim.x + threadIdx.x;
	k = blockIdx.y * blockDim.y + threadIdx.y;

	if(j < g->dim_g[1] && k < g->dim_g[2]){
		FOR(l, 0, g->nc){
			FOR(i, 0, g->ng){
				d_ptr(l,i,j,k) 					= d_ptr(l,i+g->dim[0],j,k);
				d_ptr(l,i+g->dim[0]+g->ng,j,k)	= d_ptr(l,i+g->ng,j,k);
			}
		}
	}
}
#undef	d_ptr

void gpu_fill_boundary(
	global_const_t &h_const,	// i:	Global Constants
	global_const_t *d_const,	// i:	Device Pointer to Global Constants
	double *d_ptr		 		// i/o: Device Pointer
){

	dim3 block_dim(16, 16);
	dim3 grid_dim(CEIL(h_const.dim[0], 16), CEIL(h_const.dim[1], 16));

	gpu_fill_boundary_z_kernel<<<grid_dim, block_dim>>>(d_const, d_ptr);

	grid_dim.y = CEIL(h_const.dim_g[2], 16);
	gpu_fill_boundary_y_kernel<<<grid_dim, block_dim>>>(d_const, d_ptr);

	grid_dim.x = CEIL(h_const.dim_g[1], 16);
	gpu_fill_boundary_x_kernel<<<grid_dim, block_dim>>>(d_const, d_ptr);
}

void allocate_4D(double ****&ptr, int dim[], int dl){

	int l,i,j;
	int di=dim[0], dj=dim[1], dk=dim[2];
	double *temp;

	ptr = (double ****) malloc(dl * sizeof(double ***));
	FOR(l, 0, dl){
		ptr[l] = (double ***) malloc(di * sizeof(double **));
		FOR(i, 0, di)
			ptr[l][i] = (double **) malloc(dj * sizeof(double *));
	}

	temp = (double *) malloc(dl*di*dj*dk * sizeof(double));
	FOR(l, 0, dl){
		FOR(i, 0, di){
			FOR(j, 0, dj){
				ptr[l][i][j] = temp;
				temp += dk;
			}
		}
	}

}

void allocate_3D(double ***&ptr, int dim[]){
	int i,j;
	int di=dim[0], dj=dim[1], dk=dim[2];
	double *temp;

	ptr = (double ***) malloc(di * sizeof(double **));
	FOR(i, 0, di){
		ptr[i] = (double **) malloc(dj * sizeof(double *));
	}

	// Allocate memory as a bulk
	temp = (double *) malloc(di*dj*dk * sizeof(double));
	FOR(i, 0, di){
		FOR(j, 0, dj){
			ptr[i][j] = temp;
			temp += dk;
		}
	}
}

void free_4D(double ****ptr, int dim[], int dl){
	int i,l;
	int di=dim[0], dj=dim[1];

	free(ptr[0][0][0]);
	FOR(l, 0, dl){
		FOR(i, 0, di)
			free(ptr[l][i]);
		free(ptr[l]);
	}
	free(ptr);
}

void free_3D(double ***ptr, int dim[]){
	int i;
	free(ptr[0][0]);
	FOR(i, 0, dim[0])
		free(ptr[i]);
	free(ptr);
}

void read_3D(FILE *f, double ****ptr, int dim[], int l){
	int i,j,k;
	FOR(k, 0, dim[2]){
		FOR(j, 0, dim[1]){
			FOR(i, 0, dim[0])
				fscanf(f, "%le", &ptr[l][i][j][k]);
//				fscanf(f, "%le", &ptr[i][j][k][l]);
		}
	}
}


void check_double(double a, double b, const char *name){
	if(!FEQ(a, b)){
		printf("%s = %le != %le = %s2\n", name, a, b, name);
		exit(1);
	}
}

void check_lo_hi_ng_dx( int lo[],  int hi[],  int ng,  double dx[],
									  int lo2[], int hi2[], int ng2, double dx2[] ){
	int i;
	FOR(i, 0, 3){

		if(lo[i] != lo2[i]+ng){
			printf("lo[%d] = %d != %d = lo2[%d]\n", i, lo[i], lo2[i], i);
			exit(1);
		}
		if(hi[i] != hi2[i]+ng){
			printf("hi[%d] = %d != %d = hi2[%d]\n", i, hi[i], hi2[i], i);
			exit(1);
		}
		if(!FEQ(dx[i], dx2[i])){
			printf("dx[%d] = %le != %le = dx2[%d]\n", i, dx[i], dx2[i], i);
			exit(1);
		}
	}
	if(ng != ng2){
		printf("ng = %d != %d = ng2\n", ng, ng2);
		exit(1);
	}
}

void check_4D_array( const char *name, double ****a, double ****a2, int dim[],  int la){

	int i,j,k,l;
	int exp, exp2;
	double sig, sig2;
	FOR(l, 0, la){
		FOR(i, 0, dim[0]){
			FOR(j, 0, dim[1]){
				FOR(k, 0, dim[2]){
					if(!FEQ(a[l][i][j][k], a2[l][i][j][k])){
						printf("%s[%d][%d][%d][%d] = %le != %le = %s2[%d][%d][%d][%d]\n",
								name, l, i, j, k, a[l][i][j][k], a2[l][i][j][k], name, l, i, j, k);
						printf("diff = %le\n", a[l][i][j][k] - a2[l][i][j][k]);
						sig = frexp(a[l][i][j][k], &exp);
						sig2 = frexp(a2[l][i][j][k], &exp2);
						if(exp!=exp2){
							printf("exp = %d != %d = exp2\n", exp, exp2);
							printf("sig1 = %le, sig2 = %le\n", sig, sig2);
							exit(1);
						}
						if(!FEQ(sig, sig2)){
							printf("sig = %le != %le = sig2\n", sig, sig2);
							printf("diff = %le\n", sig - sig2);
							exit(1);
						}
					}
				}
			}
		}
	}
}

void fill_boundary(
	double ****U,	// Array
	int dim[],		// Dimensions (ghost cells excluded)
	int dim_ng[]	// Dimensions (ghost cells included)
){
	int i, j, k, l;
	FOR(l, 0, NC){
		FOR(i, NG, dim[0]+NG){
			FOR(j, NG, dim[1]+NG){
				FOR(k, 0, NG){
					U[l][i][j][k] = U[l][i][j][k+dim[2]];
					U[l][i][j][k+dim[2]+NG] = U[l][i][j][k+NG];
				}
			}
		}
	}

	FOR(l, 0, NC){
		FOR(i, NG, dim[0]+NG){
			FOR(j, 0, NG){
				FOR(k, 0, dim_ng[2]){
					U[l][i][j][k] = U[l][i][j+dim[1]][k];
					U[l][i][j+dim[1]+NG][k] = U[l][i][j+NG][k];
				}
			}
		}
	}

	FOR(l, 0, NC){
		FOR(i, 0, NG){
			FOR(j, 0, dim_ng[1]){
				FOR(k, 0, dim_ng[2]){
					U[l][i][j][k] = U[l][i+dim[0]][j][k];
					U[l][i+dim[0]+NG][j][k] = U[l][i+NG][j][k];
				}
			}
		}
	}
}

void fill_boundary_test(
	global_const_t h_const, // i: Global struct containing application parameters
	global_const_t *d_const	// i: Device pointer to global struct containing application paramters
){
	int i, l, n;
	int nc, dim[3], dim_g[3];
	double dt, dt2, dx[DIM], cfl, eta, alam;
	double ****U, ****U2;
	double *d_u;
	FILE *fin, *fout;

	nc = h_const.nc;
	dim[0] = dim[1] = dim[2] = h_const.ncells;
	dim_g[0] = dim_g[1] = dim_g[2] = h_const.ncells+h_const.ng+h_const.ng;

	// Allocation
	allocate_4D(U, dim_g, nc);
	allocate_4D(U2, dim_g, nc);
	gpu_allocate_4D(d_u, dim_g, 5);

	// Initiation
	fin = fopen("../testcases/advance_input", "r");
	FOR(l, 0, nc)
		read_3D(fin, U, dim_g, l);

	fscanf(fin, "%le", &dt);
	FOR(i, 0, 3)
		fscanf(fin, "%le", &dx[i]);
	fscanf(fin, "%le", &cfl);
	fscanf(fin, "%le", &eta);
	fscanf(fin, "%le", &alam);
	fclose(fin);

	gpu_copy_from_host_4D(d_u, U, dim_g, 5);

	printf("Applying fill_boundary()...\n");
//	fill_boundary(U, dim, dim_g);
	gpu_fill_boundary(h_const, d_const, d_u);

	gpu_copy_to_host_4D(U, d_u, dim_g, 5);
	fout=fopen("../testcases/fill_boundary_output", "r");
	FOR(l, 0, nc)
		read_3D(fout, U2, dim_g, l);
	check_4D_array("U", U, U2, dim_g, nc);

	fclose(fout);
	printf("Correct!\n");

	// Free memory
	free_4D(U, dim_g, nc);
	free_4D(U2, dim_g, nc);
	gpu_free_4D(d_u);
}

void read_configurations(global_const_t &h_const, global_const_t *d_const_ptr){
	int i;
	char *dest;
	FILE *fin = fopen("../testcases/general_input", "r");

	fscanf(fin, "%d", &h_const.ng);
	fscanf(fin, "%d", &h_const.nc);
	fscanf(fin, "%d", &h_const.ncells);
	fscanf(fin, "%d %d %d", &h_const.lo[0], &h_const.lo[1], &h_const.lo[2]);
	fscanf(fin, "%d %d %d", &h_const.hi[0], &h_const.hi[1], &h_const.hi[2]);
	fscanf(fin, "%le %le %le", &h_const.dx[0], &h_const.dx[1], &h_const.dx[2]);
	fscanf(fin, "%le", &h_const.cfl);
	fscanf(fin, "%le", &h_const.eta);
	fscanf(fin, "%le", &h_const.alam);
	fscanf(fin, "%d", &h_const.nsteps);
	fscanf(fin, "%le", &h_const.dt);
	fclose(fin);

	FOR(i, 0, 3){
		h_const.dim[i] 		= h_const.hi[i] - h_const.lo[i] + 1;
		h_const.dim_g[i] 	= h_const.hi[i] - h_const.lo[i] + 1 + h_const.ng + h_const.ng;
	}
	h_const.comp_offset_g  = h_const.dim_g[0] * h_const.dim_g[1] * h_const.dim_g[2];
	h_const.comp_offset    = h_const.dim[0]   * h_const.dim[1]   * h_const.dim[2];
	h_const.plane_offset_g = h_const.dim_g[1] * h_const.dim_g[2];
	h_const.plane_offset   = h_const.dim[1]   * h_const.dim[2];

	FOR(i, 0, 3)
		h_const.dxinv[i] = 1.0E0/h_const.dx[i];

	h_const.ALP	=  0.8E0;
	h_const.BET	= -0.2E0;
	h_const.GAM	=  4.0E0/105.0E0;
	h_const.DEL	= -1.0E0/280.0E0;

	h_const.OneThird		= 1.0E0/3.0E0;
	h_const.TwoThirds		= 2.0E0/3.0E0;
	h_const.FourThirds		= 4.0E0/3.0E0;
	h_const.OneQuarter    	= 1.E0/4.E0;
    h_const.ThreeQuarters 	= 3.E0/4.E0;

	h_const.CENTER		= -205.0E0/72.0E0;
	h_const.OFF1		=  8.0E0/5.0E0;
	h_const.OFF2 		= -0.2E0;
	h_const.OFF3		=  8.0E0/315.0E0;
	h_const.OFF4		= -1.0E0/560.0E0;

	hipMemcpy(d_const_ptr, &h_const, sizeof(global_const_t), hipMemcpyHostToDevice);

	dest = (char *)d_const_ptr + ((char *)&h_const.lo - (char *)&h_const);
	hipMemcpy((int *) dest, h_const.lo, 3*sizeof(int), hipMemcpyHostToDevice);
	dest = (char *)d_const_ptr + ((char *)&h_const.hi - (char *)&h_const);
	hipMemcpy((int *) dest, h_const.hi, 3*sizeof(int), hipMemcpyHostToDevice);
	dest = (char *)d_const_ptr + ((char *)&h_const.dim - (char *)&h_const);
	hipMemcpy((int *) dest, h_const.dim, 3*sizeof(int), hipMemcpyHostToDevice);
	dest = (char *)d_const_ptr + ((char *)&h_const.dim_g - (char *)&h_const);
	hipMemcpy((int *) dest, h_const.dim_g, 3*sizeof(int), hipMemcpyHostToDevice);

	dest = (char *)d_const_ptr + ((char *)&h_const.dx - (char *)&h_const);
	hipMemcpy((int *) dest, h_const.dx, 3*sizeof(double), hipMemcpyHostToDevice);
	dest = (char *)d_const_ptr + ((char *)&h_const.dxinv - (char *)&h_const);
	hipMemcpy((int *) dest, h_const.dxinv, 3*sizeof(double), hipMemcpyHostToDevice);
}

void allocate_variables(
	double ****&U, double ****&Unew, double ****&Q, double ****&D, double ****&F,
	double *&d_U, double *&d_Unew, double *&d_Q, double *&d_D, double *&d_F
){
	int i, nc, *dim=h_const.dim, *dim_g=h_const.dim_g;
	char *dest;

	nc = h_const.nc;
	allocate_4D(U,  	dim_g, 	nc);
	allocate_4D(Unew,  	dim_g, 	nc);
	allocate_4D(Q,  	dim_g, 	nc+1);
	allocate_4D(D,  	dim, 	nc);
	allocate_4D(F, 		dim, 	nc);

	gpu_allocate_4D(d_U, 	dim_g, 	nc);
	gpu_allocate_4D(d_Unew, dim_g, 	nc);
	gpu_allocate_4D(d_Q, 	dim_g, 	nc+1);
	gpu_allocate_4D(d_D, 	dim, 	nc);
	gpu_allocate_4D(d_F, 	dim, 	nc);

	dest = (char *)d_const_ptr + ((char *)&h_const.temp - (char *)&h_const);
	FOR(i, 0, MAX_TEMP)
		gpu_allocate_3D(h_const.temp[i], dim_g);
	hipMemcpy((double *) dest, h_const.temp, MAX_TEMP*sizeof(double *), hipMemcpyHostToDevice);
}

void free_variables(
	double ****U, double ****Unew, double ****Q, double ****D, double ****F,
	double *d_U, double *d_Unew, double *d_Q, double *d_D, double *d_F
){
	int i, nc, *dim=h_const.dim, *dim_g=h_const.dim_g;
	nc = h_const.nc;

	free_4D(U,  	dim_g, 	nc);
	free_4D(Unew,  	dim_g, 	nc);
	free_4D(Q,  	dim_g, 	nc+1);
	free_4D(D,  	dim, 	nc);
	free_4D(F, 		dim, 	nc);

	gpu_free_4D(d_U);
	gpu_free_4D(d_Unew);
	gpu_free_4D(d_Q);
	gpu_free_4D(d_D);
	gpu_free_4D(d_F);

	FOR(i, 0, MAX_TEMP)
		gpu_free_3D(h_const.temp[i]);
}

void print_4D(FILE *f, double ****ptr, int dim[], int dl){
	int i,j,k,l;
	FOR(l, 0, dl){
		FOR(k, 0, dim[2]){
			FOR(j, 0, dim[1]){
				FOR(i, 0, dim[0])
					fprintf(f, "%.17e\t", ptr[l][i][j][k]);
				fprintf(f, "\n");
			}
			fprintf(f, "\n");
		}
		fprintf(f, "\n");
	}
}
