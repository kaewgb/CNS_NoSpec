#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void fill(hipPitchedPtr U){

}
int main(int argc, char *argv[]){

	struct hipPitchedPtr U;
	struct hipExtent ext;

	ext.width = 40;
	ext.height = 40;
	ext.depth = 40;
	printf("%15s pitch xsize ysize\n", "extent");

	for(ext.depth=500; ext.depth < 700; ext.depth+=100){
		for(ext.height=500; ext.height < 700; ext.height+=100){
			for(ext.width=500; ext.width < 700; ext.width+=100){
				hipMalloc3D(&U, ext);
				printf("[%3d][%3d][%3d] %5d %5d %5d\n", ext.depth, ext.height, ext.width, U.pitch, U.xsize, U.ysize);
				hipFree(U.ptr);
			}
		}
	}
	return 0;
}
