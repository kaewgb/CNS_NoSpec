#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include "header.h"
#include "util.h"
#define CUDA_SAFE_CALL( call )                               						\
{                                                              						\
    hipError_t err = call;                                                       	\
    if( hipSuccess != err) {                                                    	\
		fprintf(stderr, "Cuda error in call at file '%s' in line %i : %s.\n",     	\
				__FILE__, __LINE__, hipGetErrorString( err) );             		\
		exit(-1);                                                             		\
	}                                         										\
}

extern global_const_t h_const;
extern global_const_t *d_const_ptr;

void gpu_allocate_3D(double *&d_ptr, int dim[]){
	CUDA_SAFE_CALL(hipMalloc((void **) &d_ptr, dim[0]*dim[1]*dim[2] * sizeof(double)));
}
void gpu_copy_from_host_3D(double *dev, double ***host, int dim[]){
	CUDA_SAFE_CALL(hipMemcpy(dev, host[0][0], dim[0]*dim[1]*dim[2] * sizeof(double), hipMemcpyHostToDevice));
}
void gpu_copy_to_host_3D(double ***host, double *dev, int dim[]){
	CUDA_SAFE_CALL(hipMemcpy(host[0][0], dev, dim[0]*dim[1]*dim[2] * sizeof(double), hipMemcpyDeviceToHost));
}
void gpu_free_3D(double *d_ptr){
	CUDA_SAFE_CALL(hipFree(d_ptr));
}

void gpu_allocate_4D(double *&d_ptr, int dim[], int dl){
	CUDA_SAFE_CALL(hipMalloc((void **) &d_ptr, dim[0]*dim[1]*dim[2]*dl * sizeof(double)));
}

void gpu_copy_from_host_4D(double *dev, double ****host, int dim[], int dl){
	CUDA_SAFE_CALL(hipMemcpy(dev, host[0][0][0], dim[0]*dim[1]*dim[2]*dl * sizeof(double), hipMemcpyHostToDevice));
}

void gpu_copy_to_host_4D(double ****host, double *dev, int dim[], int dl){
	CUDA_SAFE_CALL(hipMemcpy(host[0][0][0], dev, dim[0]*dim[1]*dim[2]*dl * sizeof(double), hipMemcpyDeviceToHost));
}

void gpu_free_4D(double *d_ptr){
	CUDA_SAFE_CALL(hipFree(d_ptr));
}

#define d_ptr(l,i,j,k)	d_ptr[(l)*g->comp_offset_g + (i)*g->plane_offset_g + (j)*g->dim_g[2] + (k)]

__device__ kernel_const_t k_const;
__global__ void gpu_fill_boundary_z_kernel(
	global_const_t *g, 		// i: Global Constants
	double *d_ptr			// i/o:	Device Pointer
){
	int i,j,k,l;
	i = blockIdx.x * blockDim.x + threadIdx.x;
	j = blockIdx.y * blockDim.y + threadIdx.y;

	if(i < g->dim[0] && j < g->dim[1]){
		i += g->ng;
		j += g->ng;
		FOR(l, 0, g->nc){
			FOR(k, 0, g->ng){
				d_ptr(l,i,j,k) 					= d_ptr(l,i,j,k+g->dim[2]);
				d_ptr(l,i,j,k+g->dim[2]+g->ng)	= d_ptr(l,i,j,k+g->ng);
			}
		}
	}

}
__global__ void gpu_fill_boundary_y_kernel(
	global_const_t *g, 		// i: Global Constants
	double *d_ptr			// i/o:	Device Pointer
){
	int i,j,k,l;
	i = blockIdx.x * blockDim.x + threadIdx.x;
	k = blockIdx.y * blockDim.y + threadIdx.y;

	if(i < g->dim[0] && k < g->dim_g[2]){
		i += g->ng;
		FOR(l, 0, g->nc){
			FOR(j, 0, g->ng){
				d_ptr(l,i,j,k) 					= d_ptr(l,i,j+g->dim[1],k);
				d_ptr(l,i,j+g->dim[1]+g->ng,k)	= d_ptr(l,i,j+g->ng,k);
			}
		}
	}

}
__global__ void gpu_fill_boundary_x_kernel(
	global_const_t *g, 		// i: Global Constants
	double *d_ptr			// i/o:	Device Pointer
){
	int i,j,k,l;
	j = blockIdx.x * blockDim.x + threadIdx.x;
	k = blockIdx.y * blockDim.y + threadIdx.y;

	if(j < g->dim_g[1] && k < g->dim_g[2]){
		FOR(l, 0, g->nc){
			FOR(i, 0, g->ng){
				d_ptr(l,i,j,k) 					= d_ptr(l,i+g->dim[0],j,k);
				d_ptr(l,i+g->dim[0]+g->ng,j,k)	= d_ptr(l,i+g->ng,j,k);
			}
		}
	}
}
#undef	d_ptr

void gpu_fill_boundary(
	global_const_t &h_const,	// i:	Global Constants
	global_const_t *d_const,	// i:	Device Pointer to Global Constants
	double *d_ptr		 		// i/o: Device Pointer
){

	dim3 block_dim(16, 16);
	dim3 grid_dim(CEIL(h_const.dim[0], 16), CEIL(h_const.dim[1], 16));

	gpu_fill_boundary_z_kernel<<<grid_dim, block_dim>>>(d_const, d_ptr);

	grid_dim.y = CEIL(h_const.dim_g[2], 16);
	gpu_fill_boundary_y_kernel<<<grid_dim, block_dim>>>(d_const, d_ptr);

	grid_dim.x = CEIL(h_const.dim_g[1], 16);
	gpu_fill_boundary_x_kernel<<<grid_dim, block_dim>>>(d_const, d_ptr);
}

void fill_boundary_test(
	global_const_t h_const, // i: Global struct containing application parameters
	global_const_t *d_const	// i: Device pointer to global struct containing application paramters
){
	int i, l, n;
	int nc, dim[3], dim_g[3];
	double dt, dt2, dx[DIM], cfl, eta, alam;
	double ****U, ****U2;
	double *d_u;
	FILE *fin, *fout;

	nc = h_const.nc;
	dim[0] = dim[1] = dim[2] = h_const.ncells;
	dim_g[0] = dim_g[1] = dim_g[2] = h_const.ncells+h_const.ng+h_const.ng;

	// Allocation
	allocate_4D(U, dim_g, nc);
	allocate_4D(U2, dim_g, nc);
	gpu_allocate_4D(d_u, dim_g, 5);

	// Initiation
	fin = fopen("../testcases/advance_input", "r");
	FOR(l, 0, nc)
		read_3D(fin, U, dim_g, l);

	fscanf(fin, "%le", &dt);
	FOR(i, 0, 3)
		fscanf(fin, "%le", &dx[i]);
	fscanf(fin, "%le", &cfl);
	fscanf(fin, "%le", &eta);
	fscanf(fin, "%le", &alam);
	fclose(fin);

	gpu_copy_from_host_4D(d_u, U, dim_g, 5);

	printf("Applying fill_boundary()...\n");
//	fill_boundary(U, dim, dim_g);
	gpu_fill_boundary(h_const, d_const, d_u);

	gpu_copy_to_host_4D(U, d_u, dim_g, 5);
	fout=fopen("../testcases/fill_boundary_output", "r");
	FOR(l, 0, nc)
		read_3D(fout, U2, dim_g, l);
	check_4D_array("U", U, U2, dim_g, nc);

	fclose(fout);
	printf("Correct!\n");

	// Free memory
	free_4D(U, dim_g, nc);
	free_4D(U2, dim_g, nc);
	gpu_free_4D(d_u);
}

void copy_configurations(global_const_t h_const, global_const_t *d_const_ptr){

	char *dest;
	hipMemcpy(d_const_ptr, &h_const, sizeof(global_const_t), hipMemcpyHostToDevice);

	dest = (char *)d_const_ptr + ((char *)&h_const.lo - (char *)&h_const);
	hipMemcpy((int *) dest, h_const.lo, 3*sizeof(int), hipMemcpyHostToDevice);
	dest = (char *)d_const_ptr + ((char *)&h_const.hi - (char *)&h_const);
	hipMemcpy((int *) dest, h_const.hi, 3*sizeof(int), hipMemcpyHostToDevice);
	dest = (char *)d_const_ptr + ((char *)&h_const.dim - (char *)&h_const);
	hipMemcpy((int *) dest, h_const.dim, 3*sizeof(int), hipMemcpyHostToDevice);
	dest = (char *)d_const_ptr + ((char *)&h_const.dim_g - (char *)&h_const);
	hipMemcpy((int *) dest, h_const.dim_g, 3*sizeof(int), hipMemcpyHostToDevice);

	dest = (char *)d_const_ptr + ((char *)&h_const.dx - (char *)&h_const);
	hipMemcpy((int *) dest, h_const.dx, 3*sizeof(double), hipMemcpyHostToDevice);
	dest = (char *)d_const_ptr + ((char *)&h_const.dxinv - (char *)&h_const);
	hipMemcpy((int *) dest, h_const.dxinv, 3*sizeof(double), hipMemcpyHostToDevice);
}

void allocate_variables(
	double ****&U, double ****&Unew, double ****&Q, double ****&D, double ****&F,
	double *&d_U, double *&d_Unew, double *&d_Q, double *&d_D, double *&d_F,
	bool gpu // == true by default
){
	int i, nc, *dim=h_const.dim, *dim_g=h_const.dim_g;
	char *dest;

	nc = h_const.nc;
	allocate_4D(U,  	dim_g, 	nc);
	allocate_4D(Unew,  	dim_g, 	nc);
	allocate_4D(Q,  	dim_g, 	nc+1);
	allocate_4D(D,  	dim, 	nc);
	allocate_4D(F, 		dim, 	nc);

	if(gpu){
		gpu_allocate_4D(d_U, 	dim_g, 	nc);
		gpu_allocate_4D(d_Unew, dim_g, 	nc);
		gpu_allocate_4D(d_Q, 	dim_g, 	nc+1);
		gpu_allocate_4D(d_D, 	dim, 	nc);
		gpu_allocate_4D(d_F, 	dim, 	nc);

		dest = (char *)d_const_ptr + ((char *)&h_const.temp - (char *)&h_const);

		FOR(i, 0, MAX_TEMP)
			gpu_allocate_3D(h_const.temp[i], dim_g);
		hipMemcpy((double *) dest, h_const.temp, MAX_TEMP*sizeof(double *), hipMemcpyHostToDevice);
	}
	else {
		DO(i, 0, WZ)
			allocate_3D(h_const.cpu_temp[i], dim_g);
	}
}

void free_variables(
	double ****U, double ****Unew, double ****Q, double ****D, double ****F,
	double *d_U, double *d_Unew, double *d_Q, double *d_D, double *d_F,
	bool gpu // == true by default
){
	int i, nc, *dim=h_const.dim, *dim_g=h_const.dim_g;
	nc = h_const.nc;

	free_4D(U,  	dim_g, 	nc);
	free_4D(Unew,  	dim_g, 	nc);
	free_4D(Q,  	dim_g, 	nc+1);
	free_4D(D,  	dim, 	nc);
	free_4D(F, 		dim, 	nc);

	if(gpu){
		gpu_free_4D(d_U);
		gpu_free_4D(d_Unew);
		gpu_free_4D(d_Q);
		gpu_free_4D(d_D);
		gpu_free_4D(d_F);

		FOR(i, 0, MAX_TEMP)
			gpu_free_3D(h_const.temp[i]);
	}
	else {
		DO(i, 0, WZ)
			free_3D(h_const.cpu_temp[i], dim_g);
	}
}

