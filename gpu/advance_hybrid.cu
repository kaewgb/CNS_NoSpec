#include <stdio.h>
#include <stdlib.h>
#include "header.h"
#include "helper_functions.h"

void advance_hybrid(
	global_const_t &h_const,
	global_const_t *d_const,
	double ****U,	// i/o
	double *d_U,	// i/o
	double &dt,		// o
	double dx[],	// i: dx[U.dim]
	double cfl,		// i
	double eta,		// i
	double alam		// i
){
	int lo[3], hi[3], i, j, k, l, n, nc, ng;
	double courno, courno_proc;
	double ****D, ****F, ****Unew, ****Q;
	double ****Q2, ****D2, ****F2, ****Unew2, ****U2;
	double *d_Unew, *d_Q, *d_D, *d_F;

    // Some arithmetic constants.
    double OneThird      = 1.E0/3.E0;
    double TwoThirds     = 2.E0/3.E0;
    double OneQuarter    = 1.E0/4.E0;
    double ThreeQuarters = 3.E0/4.E0;

	nc = NC; // ncomp(U)
	ng = NG; // nghost(U)

	int dim[3], dim_g[3];
	dim[0] 		= dim[1] 	= dim[2] 	= NCELLS;
	dim_g[0] 	= dim_g[1]	= dim_g[2]	= NCELLS+NG+NG;

	lo[0] = lo[1] = lo[2] = NG;
	hi[0] = hi[1] = hi[2] = NCELLS-1+NG;

	// Allocation
	allocate_4D(D, dim, nc);
	allocate_4D(D2, dim, nc);
	allocate_4D(F, dim, nc);
	allocate_4D(F2, dim, nc);
	allocate_4D(Q, dim_g, nc+1);
	allocate_4D(Q2, dim_g, nc+1);
	allocate_4D(Unew, dim_g, nc);
	allocate_4D(Unew2, dim_g, nc);
	allocate_4D(U2, dim_g, nc);

	gpu_allocate_4D(d_Unew, dim_g, 	5);
	gpu_allocate_4D(d_Q, 	dim_g, 	6);
	gpu_allocate_4D(d_D, 	dim, 	5);
	gpu_allocate_4D(d_F, 	dim, 	5);

	FOR(i, 0, MAX_TEMP)
		gpu_allocate_3D(h_const.temp[i], dim_g);

	//
	// multifab_fill_boundary(U)
	//
//	gpu_copy_from_host_4D(d_U, U, dim_g, nc);
//	gpu_fill_boundary(h_const, d_const, d_U);
//	gpu_copy_to_host_4D(U, d_U, dim_g, nc);
	fill_boundary(U, dim, dim_g);

    //!
    //! Calculate primitive variables based on U.
    //!
    //! Also calculate courno so we can set "dt".
    //!
	courno_proc = 1.0E-50;
//	gpu_ctoprim(h_const, d_const, d_U, d_Q, courno_proc);
//	gpu_copy_to_host_4D(Q, d_Q, dim_g, nc+1);
	ctoprim(lo, hi, U, Q, dx, ng, courno_proc);

	courno = courno_proc;
	dt = cfl/courno;
	printf("dt, courno = %le, %le\n", dt, courno);

    //!
    //! Calculate D at time N.
    //!
//    gpu_diffterm(h_const, d_const, d_Q, d_D);
//    gpu_copy_to_host_4D(D, d_D, dim, nc);
	diffterm(lo, hi, ng, dx, Q, D, eta, alam);

    //!
    //! Calculate F at time N.
    //!
//    gpu_hypterm(h_const, d_const, d_U, d_Q, d_F);
//    gpu_copy_to_host_4D(F, d_F, dim, nc);
	hypterm(lo, hi, ng, dx, U, Q, F);

    //!
    //! Calculate U at time N+1/3.
    //!
//	FOR(l, 0, nc){
//		FOR(i, 0, dim[0]){
//			FOR(j, 0, dim[1]){
//				FOR(k, 0, dim[2]){
//					Unew[l][i+NG][j+NG][k+NG] = U[l][i+NG][j+NG][k+NG] + dt*(D[l][i][j][k] + F[l][i][j][k]);
//				}
//			}
//		}
//	}

	gpu_copy_from_host_4D(d_Unew, 	Unew, 	dim_g, 	nc);
	gpu_copy_from_host_4D(d_U, 		U, 		dim_g, 	nc);
	gpu_copy_from_host_4D(d_D, 		D, 		dim, 	nc);
	gpu_copy_from_host_4D(d_F, 		F, 		dim, 	nc);
	gpu_Unew(h_const, d_const, d_Unew, d_U, d_D, d_F, dt, 1);
	gpu_copy_to_host_4D(Unew, d_Unew, dim_g, nc);

	//!
    //! Sync U^1/3 prior to calculating D & F. -- multifab_fill_boundary(Unew)
    //!
	fill_boundary(Unew, dim, dim_g);

	//!
    //! Calculate primitive variables based on U^1/3.
    //!
	ctoprim(lo, hi, Unew, Q, dx, ng);

    //!
    //! Calculate D at time N+1/3.
    //!
	diffterm(lo, hi, ng, dx, Q, D, eta, alam);

	//!
    //! Calculate F at time N+1/3.
    //!
	hypterm(lo, hi, ng, dx, Unew, Q, F);

	//!
    //! Calculate U at time N+2/3.
    //!
//    FOR(l, 0, nc){
//		FOR(i, 0, dim[0]){
//			FOR(j, 0, dim[0]){
//				FOR(k, 0, dim[0]){
//					Unew[l][i+NG][j+NG][k+NG] =
//						ThreeQuarters *  U[l][i+NG][j+NG][k+NG] +
//						OneQuarter    * (Unew[l][i+NG][j+NG][k+NG] + dt*(D[l][i][j][k] + F[l][i][j][k]));
//				}
//			}
//		}
//    }
    gpu_copy_from_host_4D(d_Unew, 	Unew, 	dim_g, 	nc);
	gpu_copy_from_host_4D(d_U, 		U, 		dim_g, 	nc);
	gpu_copy_from_host_4D(d_D, 		D, 		dim, 	nc);
	gpu_copy_from_host_4D(d_F, 		F, 		dim, 	nc);
	gpu_Unew(h_const, d_const, d_Unew, d_U, d_D, d_F, dt, 2);
	gpu_copy_to_host_4D(Unew, d_Unew, dim_g, nc);

	//!
    //! Sync U^2/3 prior to calculating D & F. -- multifab_fill_boundary(Unew)
    //!
	fill_boundary(Unew, dim, dim_g);

    //!
    //! Calculate primitive variables based on U^2/3.
    //!
	ctoprim(lo, hi, Unew, Q, dx, ng);

    //!
    //! Calculate D at time N+2/3.
    //!
    diffterm(lo, hi, ng, dx, Q, D, eta, alam);

    //!
    //! Calculate F at time N+2/3.
    //!
	hypterm(lo, hi, ng, dx, Unew, Q, F);

    //!
    //! Calculate U at time N+1.
    //!
//    FOR(l, 0, nc){
//		FOR(i, 0, dim[0]){
//			FOR(j, 0, dim[0]){
//				FOR(k, 0, dim[0]){
//					U[l][i+NG][j+NG][k+NG] =
//						OneThird    *  U[l][i+NG][j+NG][k+NG] +
//						TwoThirds   * (Unew[l][i+NG][j+NG][k+NG] + dt*(D[l][i][j][k] + F[l][i][j][k]));
//				}
//			}
//		}
//    }
    gpu_copy_from_host_4D(d_Unew, 	Unew, 	dim_g, 	nc);
	gpu_copy_from_host_4D(d_U, 		U, 		dim_g, 	nc);
	gpu_copy_from_host_4D(d_D, 		D, 		dim, 	nc);
	gpu_copy_from_host_4D(d_F, 		F, 		dim, 	nc);
	gpu_Unew(h_const, d_const, d_Unew, d_U, d_D, d_F, dt, 3);
	gpu_copy_to_host_4D(U, d_U, dim_g, nc);

	// Free memory
	free_4D(D, 		dim, 	nc);
	free_4D(D2, 	dim,	nc);
	free_4D(F, 		dim,	nc);
	free_4D(F2, 	dim,	nc);
	free_4D(Q, 		dim_g,	nc+1);
	free_4D(Q2, 	dim_g,	nc+1);
	free_4D(Unew, 	dim_g,	nc);
	free_4D(Unew2, 	dim_g,	nc);
	free_4D(U2, 	dim_g,	nc);

	gpu_free_4D(d_Unew);
	gpu_free_4D(d_Q);
	gpu_free_4D(d_D);
	gpu_free_4D(d_F);

	FOR(i, 0, MAX_TEMP)
		gpu_free_3D(h_const.temp[i]);
}

void advance_hybrid_test(
	global_const_t &h_const, 	// i: Global struct containing application parameters
	global_const_t *d_const		// i: Device pointer to global struct containing application paramters
){
	int i, l, n;
	int nc, dim_g[3];
	double dt, dt2, dx[DIM], cfl, eta, alam;
	double ****U, ****U2;
	double *d_U;
	FILE *fin, *fout;

	nc = NC;
	dim_g[0] = dim_g[1] = dim_g[2] = NCELLS+NG+NG;

	// Allocation
	allocate_4D(U, dim_g, nc);
	allocate_4D(U2, dim_g, nc);
	gpu_allocate_4D(d_U, dim_g, 5);

	// Initiation
	fin = fopen("../testcases/advance_input", "r");
	FOR(l, 0, nc)
		read_3D(fin, U, dim_g, l);

	fscanf(fin, "%le", &dt);
	FOR(i, 0, 3)
		fscanf(fin, "%le", &dx[i]);
	fscanf(fin, "%le", &cfl);
	fscanf(fin, "%le", &eta);
	fscanf(fin, "%le", &alam);
	fclose(fin);

	advance_hybrid(h_const, d_const, U, d_U, dt, dx, cfl, eta, alam);

	fout=fopen("../testcases/advance_output", "r");
	FOR(l, 0, nc)
		read_3D(fout, U2, dim_g, l);
	check_4D_array("U", U, U2, dim_g, nc);

	fscanf(fout, "%le", &dt2);
	check_double(dt, dt2, "dt");
	fclose(fout);
	printf("Correct!\n");

	// Free memory
	free_4D(U,  dim_g, nc);
	free_4D(U2, dim_g, nc);
	gpu_free_4D(d_U);
}
