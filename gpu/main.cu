#include <stdio.h>
#include <hip/hip_runtime.h>
#include "header.h"
#include "helper_functions.h"

global_const_t h_const;
global_const_t *d_const_ptr;
__constant__ global_const_t d_const;
__constant__ kernel_const_t kc;

int main(int argc, char *argv[]){

	//!
	//! Variable Declaration
	//!
	char *dest;
	FILE *fin, *fout;
	int i, l;
	double dt;
	double ****U, ****Unew, ****Q, ****D, ****F;
	double *d_U, *d_Unew, *d_Q, *d_D, *d_F;

	hipGetSymbolAddress((void **) &d_const_ptr, d_const);
	hipGetSymbolAddress((void **) &(h_const.kc), kc);

	//!
	//! Prepare Global Constants
	//!
	read_configurations(h_const, d_const_ptr);

	//!
	//! Allocation
	//!
	allocate_variables(U, Unew, Q, D, F, d_U, d_Unew, d_Q, d_D, d_F);

	//!
	//! Advance
	//!
	fin = fopen("../testcases/multistep_input", "r");
	FOR(l, 0, h_const.nc)
		read_3D(fin, U, h_const.dim_g, l);
	fclose(fin);

	gpu_copy_from_host_4D(d_U, U, h_const.dim_g, h_const.nc);

	FOR(i, 0, h_const.nsteps)
		gpu_advance(h_const, d_const_ptr, d_U, d_Unew, d_Q, d_D, d_F, dt);

	gpu_copy_to_host_4D(U, d_U, h_const.dim_g, h_const.nc);

	fout = fopen("output", "w");
	fprintf(fout, "%d\n", h_const.nc);
	fprintf(fout, "%d %d %d\n", h_const.dim_g[0], h_const.dim_g[1], h_const.dim_g[2]);
	print_4D(fout, U, h_const.dim_g, h_const.nc);
	fclose(fout);

	//!
	//!	Free Allocations
	//!
	free_variables(U, Unew, Q, D, F, d_U, d_Unew, d_Q, d_D, d_F);

	return 0;

}

