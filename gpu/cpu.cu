#include <stdio.h>
#include <hip/hip_runtime.h>
#include "header.h"
#include "util.h"

global_const_t h_const;
global_const_t *d_const_ptr;
__constant__ global_const_t d_const;
__constant__ kernel_const_t kc;

int main(int argc, char *argv[]){

	//!
	//! Variable Declaration
	//!
	char *dest;
	FILE *fin, *fout;
	int i, l;
	double dt, total_time;
	double ****U, ****Unew, ****Q, ****D, ****F;
	double *d_U, *d_Unew, *d_Q, *d_D, *d_F;

	hipGetSymbolAddress((void **) &d_const_ptr, d_const);
	hipGetSymbolAddress((void **) &(h_const.kc), kc);

	//!
	//! Prepare Global Constants
	//!
	read_configurations(h_const, argc, argv);
	copy_configurations(h_const, d_const_ptr);

	//!
	//! Allocation
	//!
	allocate_variables(U, Unew, Q, D, F, d_U, d_Unew, d_Q, d_D, d_F, false, true);

	//!
	//! Advance
	//!
	fin = fopen(h_const.input_file_name, "r");
	FOR(l, 0, h_const.nc)
		read_3D(fin, U, h_const.dim_g, l);
	fclose(fin);
//	init_data(h_const, U);

	total_time = -get_time();
	FOR(i, 0, h_const.nsteps)
		advance(h_const, U, Unew, Q, D, F, dt);

	total_time += get_time();
	printf("Total time: %lf\n", total_time);


	fout = fopen("correct", "w");
	fprintf(fout, "%d\n", h_const.nc);
	fprintf(fout, "%d %d %d\n", h_const.dim_g[0], h_const.dim_g[1], h_const.dim_g[2]);
	print_4D(fout, U, h_const.dim_g, h_const.nc);
	fclose(fout);


	//!
	//!	Free Allocations
	//!
	free_variables(U, Unew, Q, D, F, d_U, d_Unew, d_Q, d_D, d_F, false);

	return 0;

}

