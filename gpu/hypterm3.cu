#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "header.h"
#include "util.cuh"
#include "util.h"

#define BLOCK_SMALL		8
#define	BLOCK_LARGE		16
#define	THREAD_Z		8

__global__ void gpu_hypterm_xy_stencil_kernel(
	global_const_t *g,	// i:
	double *cons,		// i:
	double *q,			// i:
	double *flux		// o:
){
	bool compute=false;
	int z,idx,out,si,sj,sk,tidx,tidy;
	double unp1, unp2, unp3, unp4, unm1, unm2, unm3, unm4;
	double flux_irho, flux_imx, flux_imy, flux_imz, flux_iene;

	__shared__ double      s_qu[BLOCK_LARGE+NG+NG][BLOCK_LARGE+NG+NG];
	__shared__ double      s_qv[BLOCK_LARGE+NG+NG][BLOCK_LARGE+NG+NG];
	__shared__ double   s_qpres[BLOCK_LARGE+NG+NG][BLOCK_LARGE+NG+NG];
	__shared__ double    s_cons[BLOCK_LARGE+NG+NG][BLOCK_LARGE+NG+NG];

	// Load to shared mem
	for(z=0;z<THREAD_Z;z++){

		si = blockIdx.x*blockDim.x+threadIdx.x;
		sj = blockIdx.y*blockDim.y+threadIdx.y;
		sk = (blockIdx.z*blockDim.z+threadIdx.z)*THREAD_Z + z;

		out = sk*g->plane_offset_padded + sj*g->pitch[0] + si;
		compute = (si < g->dim[0] && sj < g->dim[1] && sk < g->dim[2]);

		for(sj=blockIdx.y*blockDim.y+threadIdx.y, tidy=threadIdx.y; tidy < BLOCK_LARGE+NG+NG; sj+=blockDim.y, tidy+=blockDim.y){
			for(si=blockIdx.x*blockDim.x+threadIdx.x, tidx=threadIdx.x; tidx < BLOCK_LARGE+NG+NG; si+=blockDim.x, tidx+=blockDim.x){
				if( si < g->dim_g[0] && sj < g->dim_g[1] && sk < g->dim_g[2]){

					idx = (sk+g->ng)*g->plane_offset_g_padded + sj*g->pitch_g[0] + si;

							  s_qu[tidy][tidx]  =     q[idx + qu*g->comp_offset_g_padded];
							  s_qv[tidy][tidx]  =     q[idx + qv*g->comp_offset_g_padded];
						   s_qpres[tidy][tidx]	=     q[idx + qpres*g->comp_offset_g_padded];
							s_cons[tidy][tidx] 	=  cons[idx + imx*g->comp_offset_g_padded];
				}
			}
		}
		__syncthreads();

		if(compute){

	#define	s_qu(i)			s_qu[threadIdx.y+g->ng][threadIdx.x+g->ng+(i)]
	#define	s_qv(i)			s_qv[threadIdx.y+g->ng+(i)][threadIdx.x+g->ng]
	#define	s_qpres_x(i)	s_qpres[threadIdx.y+g->ng][threadIdx.x+g->ng+(i)]
	#define	s_qpres_y(i)	s_qpres[threadIdx.y+g->ng+(i)][threadIdx.x+g->ng]
	#define	s_imx_x(i)	    s_cons[threadIdx.y+g->ng][threadIdx.x+g->ng+(i)]
	#define	s_imx_y(i)	    s_cons[threadIdx.y+g->ng+(i)][threadIdx.x+g->ng]

			flux_irho = - ( g->ALP*(s_imx_x(1)-s_imx_x(-1))
						  + g->BET*(s_imx_x(2)-s_imx_x(-2))
						  + g->GAM*(s_imx_x(3)-s_imx_x(-3))
						  + g->DEL*(s_imx_x(4)-s_imx_x(-4)))*g->dxinv[0];

			flux_imx  = - ( g->ALP*(s_imx_x(1)*s_qu(1)-s_imx_x(-1)*s_qu(-1)
						  + (s_qpres_x(1)-s_qpres_x(-1)))
						  + g->BET*(s_imx_x(2)*s_qu(2)-s_imx_x(-2)*s_qu(-2)
						  + (s_qpres_x(2)-s_qpres_x(-2)))
						  + g->GAM*(s_imx_x(3)*s_qu(3)-s_imx_x(-3)*s_qu(-3)
						  + (s_qpres_x(3)-s_qpres_x(-3)))
						  + g->DEL*(s_imx_x(4)*s_qu(4)-s_imx_x(-4)*s_qu(-4)
						  + (s_qpres_x(4)-s_qpres_x(-4))))*g->dxinv[0];

			flux_imx -=   ( g->ALP*(s_imx_y(1)*s_qv(1)-s_imx_y(-1)*s_qv(-1))
						  + g->BET*(s_imx_y(2)*s_qv(2)-s_imx_y(-2)*s_qv(-2))
						  + g->GAM*(s_imx_y(3)*s_qv(3)-s_imx_y(-3)*s_qv(-3))
						  + g->DEL*(s_imx_y(4)*s_qv(4)-s_imx_y(-4)*s_qv(-4)))*g->dxinv[1];

		}
	#undef s_imx_x
	#undef s_imx_y
	#define	s_imy_x(i)	    s_cons[threadIdx.y+g->ng][threadIdx.x+g->ng+(i)]
	#define	s_imy_y(i)	    s_cons[threadIdx.y+g->ng+(i)][threadIdx.x+g->ng]

		__syncthreads();
		for(sj=blockIdx.y*blockDim.y+threadIdx.y, tidy=threadIdx.y; tidy < BLOCK_LARGE+NG+NG; sj+=blockDim.y, tidy+=blockDim.y){
			for(si=blockIdx.x*blockDim.x+threadIdx.x, tidx=threadIdx.x; tidx < BLOCK_LARGE+NG+NG; si+=blockDim.x, tidx+=blockDim.x){
				if( si < g->dim_g[0] && sj < g->dim_g[1] && sk < g->dim_g[2]){

					idx = (sk+g->ng)*g->plane_offset_g_padded + sj*g->pitch_g[0] + si;
					s_cons[tidy][tidx] 	=  cons[idx + imy*g->comp_offset_g_padded];
				}
			}
		}
		__syncthreads();

		if(compute){

			flux_irho -=  ( g->ALP*(s_imy_y(1)-s_imy_y(-1))
						  + g->BET*(s_imy_y(2)-s_imy_y(-2))
						  + g->GAM*(s_imy_y(3)-s_imy_y(-3))
						  + g->DEL*(s_imy_y(4)-s_imy_y(-4)))*g->dxinv[1];

			flux_imy =   -( g->ALP*(s_imy_y(1)*s_qv(1)-s_imy_y(-1)*s_qv(-1)
						  + (s_qpres_y(1)-s_qpres_y(-1)))
						  + g->BET*(s_imy_y(2)*s_qv(2)-s_imy_y(-2)*s_qv(-2)
						  + (s_qpres_y(2)-s_qpres_y(-2)))
						  + g->GAM*(s_imy_y(3)*s_qv(3)-s_imy_y(-3)*s_qv(-3)
						  + (s_qpres_y(3)-s_qpres_y(-3)))
						  + g->DEL*(s_imy_y(4)*s_qv(4)-s_imy_y(-4)*s_qv(-4)
						  + (s_qpres_y(4)-s_qpres_y(-4))))*g->dxinv[1];

			flux_imy  -=  ( g->ALP*(s_imy_x(1)*s_qu(1)-s_imy_x(-1)*s_qu(-1))
						  + g->BET*(s_imy_x(2)*s_qu(2)-s_imy_x(-2)*s_qu(-2))
						  + g->GAM*(s_imy_x(3)*s_qu(3)-s_imy_x(-3)*s_qu(-3))
						  + g->DEL*(s_imy_x(4)*s_qu(4)-s_imy_x(-4)*s_qu(-4)))*g->dxinv[0];

		}
	#undef  s_imy_x
	#undef  s_imy_y
	#define	s_imz_x(i)	    s_cons[threadIdx.y+g->ng][threadIdx.x+g->ng+(i)]
	#define	s_imz_y(i)	    s_cons[threadIdx.y+g->ng+(i)][threadIdx.x+g->ng]

		__syncthreads();
		for(sj=blockIdx.y*blockDim.y+threadIdx.y, tidy=threadIdx.y; tidy < BLOCK_LARGE+NG+NG; sj+=blockDim.y, tidy+=blockDim.y){
			for(si=blockIdx.x*blockDim.x+threadIdx.x, tidx=threadIdx.x; tidx < BLOCK_LARGE+NG+NG; si+=blockDim.x, tidx+=blockDim.x){
				if( si < g->dim_g[0] && sj < g->dim_g[1] && sk < g->dim_g[2]){

					idx = (sk+g->ng)*g->plane_offset_g_padded + sj*g->pitch_g[0] + si;
					s_cons[tidy][tidx] 	=  cons[idx + imz*g->comp_offset_g_padded];
				}
			}
		}
		__syncthreads();

		if(compute){

			flux_imz  = - ( g->ALP*(s_imz_x(1)*s_qu(1)-s_imz_x(-1)*s_qu(-1))
						  + g->BET*(s_imz_x(2)*s_qu(2)-s_imz_x(-2)*s_qu(-2))
						  + g->GAM*(s_imz_x(3)*s_qu(3)-s_imz_x(-3)*s_qu(-3))
						  + g->DEL*(s_imz_x(4)*s_qu(4)-s_imz_x(-4)*s_qu(-4)))*g->dxinv[0];

			flux_imz -=   ( g->ALP*(s_imz_y(1)*s_qv(1)-s_imz_y(-1)*s_qv(-1))
						  + g->BET*(s_imz_y(2)*s_qv(2)-s_imz_y(-2)*s_qv(-2))
						  + g->GAM*(s_imz_y(3)*s_qv(3)-s_imz_y(-3)*s_qv(-3))
						  + g->DEL*(s_imz_y(4)*s_qv(4)-s_imz_y(-4)*s_qv(-4)))*g->dxinv[1];

		}

	#undef  s_imz_x
	#undef  s_imz_y
	#define	s_iene_x(i)	    s_cons[threadIdx.y+g->ng][threadIdx.x+g->ng+(i)]
	#define	s_iene_y(i)	    s_cons[threadIdx.y+g->ng+(i)][threadIdx.x+g->ng]

		__syncthreads();
		for(sj=blockIdx.y*blockDim.y+threadIdx.y, tidy=threadIdx.y; tidy < BLOCK_LARGE+NG+NG; sj+=blockDim.y, tidy+=blockDim.y){
			for(si=blockIdx.x*blockDim.x+threadIdx.x, tidx=threadIdx.x; tidx < BLOCK_LARGE+NG+NG; si+=blockDim.x, tidx+=blockDim.x){
				if( si < g->dim_g[0] && sj < g->dim_g[1] && sk < g->dim_g[2]){

					idx = (sk+g->ng)*g->plane_offset_g_padded + sj*g->pitch_g[0] + si;
					s_cons[tidy][tidx] 	=  cons[idx + iene*g->comp_offset_g_padded];
				}
			}
		}
		__syncthreads();

		if(compute){

			flux_iene =  -( g->ALP*(s_iene_x(1)*s_qu(1)-s_iene_x(-1)*s_qu(-1)
						  + (s_qpres_x(1)*s_qu(1)-s_qpres_x(-1)*s_qu(-1)))
						  + g->BET*(s_iene_x(2)*s_qu(2)-s_iene_x(-2)*s_qu(-2)
						  + (s_qpres_x(2)*s_qu(2)-s_qpres_x(-2)*s_qu(-2)))
						  + g->GAM*(s_iene_x(3)*s_qu(3)-s_iene_x(-3)*s_qu(-3)
						  + (s_qpres_x(3)*s_qu(3)-s_qpres_x(-3)*s_qu(-3)))
						  + g->DEL*(s_iene_x(4)*s_qu(4)-s_iene_x(-4)*s_qu(-4)
						  + (s_qpres_x(4)*s_qu(4)-s_qpres_x(-4)*s_qu(-4))))*g->dxinv[0];

			flux_iene -=  ( g->ALP*(s_iene_y(1)*s_qv(1)-s_iene_y(-1)*s_qv(-1)
						  + (s_qpres_y(1)*s_qv(1)-s_qpres_y(-1)*s_qv(-1)))
						  + g->BET*(s_iene_y(2)*s_qv(2)-s_iene_y(-2)*s_qv(-2)
						  + (s_qpres_y(2)*s_qv(2)-s_qpres_y(-2)*s_qv(-2)))
						  + g->GAM*(s_iene_y(3)*s_qv(3)-s_iene_y(-3)*s_qv(-3)
						  + (s_qpres_y(3)*s_qv(3)-s_qpres_y(-3)*s_qv(-3)))
						  + g->DEL*(s_iene_y(4)*s_qv(4)-s_iene_y(-4)*s_qv(-4)
						  + (s_qpres_y(4)*s_qv(4)-s_qpres_y(-4)*s_qv(-4))))*g->dxinv[1];
		}

	#undef  s_iene_x
	#undef  s_iene_y
	#undef	s_qu
	#undef  s_qv
	#undef 	s_qpres_x
	#undef 	s_qpres_y


		/** Z dimension **/
		si = blockIdx.x*blockDim.x+threadIdx.x;
		sj = blockIdx.y*blockDim.y+threadIdx.y;
		if(compute){

			idx = (sk+g->ng)*g->plane_offset_g_padded + (sj+g->ng)*g->pitch_g[0] + si+g->ng;

			double imxp1, imxp2, imxp3, imxp4;
			double imxm1, imxm2, imxm3, imxm4;
			double imyp1, imyp2, imyp3, imyp4;
			double imym1, imym2, imym3, imym4;
			double imzp1, imzp2, imzp3, imzp4;
			double imzm1, imzm2, imzm3, imzm4;
			double ienep1, ienep2, ienep3, ienep4;
			double ienem1, ienem2, ienem3, ienem4;
			double qpresp1, qpresp2, qpresp3, qpresp4;
			double qpresm1, qpresm2, qpresm3, qpresm4;

			unp1 = q[qw*g->comp_offset_g_padded + idx + 1*g->plane_offset_g_padded];
			unp2 = q[qw*g->comp_offset_g_padded + idx + 2*g->plane_offset_g_padded];
			unp3 = q[qw*g->comp_offset_g_padded + idx + 3*g->plane_offset_g_padded];
			unp4 = q[qw*g->comp_offset_g_padded + idx + 4*g->plane_offset_g_padded];
			unm1 = q[qw*g->comp_offset_g_padded + idx - 1*g->plane_offset_g_padded];
			unm2 = q[qw*g->comp_offset_g_padded + idx - 2*g->plane_offset_g_padded];
			unm3 = q[qw*g->comp_offset_g_padded + idx - 3*g->plane_offset_g_padded];
			unm4 = q[qw*g->comp_offset_g_padded + idx - 4*g->plane_offset_g_padded];

			imxp1 = cons[imx*g->comp_offset_g_padded + idx + 1*g->plane_offset_g_padded];
			imxp2 = cons[imx*g->comp_offset_g_padded + idx + 2*g->plane_offset_g_padded];
			imxp3 = cons[imx*g->comp_offset_g_padded + idx + 3*g->plane_offset_g_padded];
			imxp4 = cons[imx*g->comp_offset_g_padded + idx + 4*g->plane_offset_g_padded];
			imxm1 = cons[imx*g->comp_offset_g_padded + idx - 1*g->plane_offset_g_padded];
			imxm2 = cons[imx*g->comp_offset_g_padded + idx - 2*g->plane_offset_g_padded];
			imxm3 = cons[imx*g->comp_offset_g_padded + idx - 3*g->plane_offset_g_padded];
			imxm4 = cons[imx*g->comp_offset_g_padded + idx - 4*g->plane_offset_g_padded];

			imyp1 = cons[imy*g->comp_offset_g_padded + idx + 1*g->plane_offset_g_padded];
			imyp2 = cons[imy*g->comp_offset_g_padded + idx + 2*g->plane_offset_g_padded];
			imyp3 = cons[imy*g->comp_offset_g_padded + idx + 3*g->plane_offset_g_padded];
			imyp4 = cons[imy*g->comp_offset_g_padded + idx + 4*g->plane_offset_g_padded];
			imym1 = cons[imy*g->comp_offset_g_padded + idx - 1*g->plane_offset_g_padded];
			imym2 = cons[imy*g->comp_offset_g_padded + idx - 2*g->plane_offset_g_padded];
			imym3 = cons[imy*g->comp_offset_g_padded + idx - 3*g->plane_offset_g_padded];
			imym4 = cons[imy*g->comp_offset_g_padded + idx - 4*g->plane_offset_g_padded];

			imzp1 = cons[imz*g->comp_offset_g_padded + idx + 1*g->plane_offset_g_padded];
			imzp2 = cons[imz*g->comp_offset_g_padded + idx + 2*g->plane_offset_g_padded];
			imzp3 = cons[imz*g->comp_offset_g_padded + idx + 3*g->plane_offset_g_padded];
			imzp4 = cons[imz*g->comp_offset_g_padded + idx + 4*g->plane_offset_g_padded];
			imzm1 = cons[imz*g->comp_offset_g_padded + idx - 1*g->plane_offset_g_padded];
			imzm2 = cons[imz*g->comp_offset_g_padded + idx - 2*g->plane_offset_g_padded];
			imzm3 = cons[imz*g->comp_offset_g_padded + idx - 3*g->plane_offset_g_padded];
			imzm4 = cons[imz*g->comp_offset_g_padded + idx - 4*g->plane_offset_g_padded];

			ienep1 = cons[iene*g->comp_offset_g_padded + idx + 1*g->plane_offset_g_padded];
			ienep2 = cons[iene*g->comp_offset_g_padded + idx + 2*g->plane_offset_g_padded];
			ienep3 = cons[iene*g->comp_offset_g_padded + idx + 3*g->plane_offset_g_padded];
			ienep4 = cons[iene*g->comp_offset_g_padded + idx + 4*g->plane_offset_g_padded];
			ienem1 = cons[iene*g->comp_offset_g_padded + idx - 1*g->plane_offset_g_padded];
			ienem2 = cons[iene*g->comp_offset_g_padded + idx - 2*g->plane_offset_g_padded];
			ienem3 = cons[iene*g->comp_offset_g_padded + idx - 3*g->plane_offset_g_padded];
			ienem4 = cons[iene*g->comp_offset_g_padded + idx - 4*g->plane_offset_g_padded];

			qpresp1 = q[qpres*g->comp_offset_g_padded + idx + 1*g->plane_offset_g_padded];
			qpresp2 = q[qpres*g->comp_offset_g_padded + idx + 2*g->plane_offset_g_padded];
			qpresp3 = q[qpres*g->comp_offset_g_padded + idx + 3*g->plane_offset_g_padded];
			qpresp4 = q[qpres*g->comp_offset_g_padded + idx + 4*g->plane_offset_g_padded];
			qpresm1 = q[qpres*g->comp_offset_g_padded + idx - 1*g->plane_offset_g_padded];
			qpresm2 = q[qpres*g->comp_offset_g_padded + idx - 2*g->plane_offset_g_padded];
			qpresm3 = q[qpres*g->comp_offset_g_padded + idx - 3*g->plane_offset_g_padded];
			qpresm4 = q[qpres*g->comp_offset_g_padded + idx - 4*g->plane_offset_g_padded];


			flux_irho -=  ( g->ALP*(imzp1-imzm1)
						  + g->BET*(imzp2-imzm2)
						  + g->GAM*(imzp3-imzm3)
						  + g->DEL*(imzp4-imzm4))*g->dxinv[2];

			flux_imx -=   ( g->ALP*(imxp1*unp1-imxm1*unm1)
						  + g->BET*(imxp2*unp2-imxm2*unm2)
						  + g->GAM*(imxp3*unp3-imxm3*unm3)
						  + g->DEL*(imxp4*unp4-imxm4*unm4))*g->dxinv[2];

			flux_imy -=   ( g->ALP*(imyp1*unp1-imym1*unm1)
						  + g->BET*(imyp2*unp2-imym2*unm2)
						  + g->GAM*(imyp3*unp3-imym3*unm3)
						  + g->DEL*(imyp4*unp4-imym4*unm4))*g->dxinv[2];

			flux_imz -=   ( g->ALP*(imzp1*unp1-imzm1*unm1
						  + (qpresp1-qpresm1))
						  + g->BET*(imzp2*unp2-imzm2*unm2
						  + (qpresp2-qpresm2))
						  + g->GAM*(imzp3*unp3-imzm3*unm3
						  + (qpresp3-qpresm3))
						  + g->DEL*(imzp4*unp4-imzm4*unm4
						  + (qpresp4-qpresm4)))*g->dxinv[2];

			flux_iene -=  ( g->ALP*(ienep1*unp1-ienem1*unm1
						  + (qpresp1*unp1-qpresm1*unm1))
						  + g->BET*(ienep2*unp2-ienem2*unm2
						  + (qpresp2*unp2-qpresm2*unm2))
						  + g->GAM*(ienep3*unp3-ienem3*unm3
						  + (qpresp3*unp3-qpresm3*unm3))
						  + g->DEL*(ienep4*unp4-ienem4*unm4
						  + (qpresp4*unp4-qpresm4*unm4)))*g->dxinv[2];

			// Update global memory
			flux[out + irho*g->comp_offset_padded] = flux_irho;
			flux[out + imx *g->comp_offset_padded] = flux_imx;
			flux[out + imy*g->comp_offset_padded] = flux_imy;
			flux[out + imz*g->comp_offset_padded] = flux_imz;
			flux[out + iene*g->comp_offset_padded] = flux_iene;
		}
	}
}

void gpu_hypterm3(
	global_const_t h_const, 	// i: Global struct containing application parameters
	global_const_t *d_const,	// i: Device pointer to global struct containing application paramters
	double *d_cons,				// i:
	double *d_q,				// i:
	double *d_flux				// o: flux
){

	/** d_flux must be set to zero beforehand (in diffterm, etc) **/

	dim3 block_dim_xy_stencil(BLOCK_LARGE, BLOCK_LARGE, 1);
	dim3 grid_dim_xy_stencil(CEIL(h_const.dim[0], BLOCK_LARGE), CEIL(h_const.dim[1], BLOCK_LARGE), CEIL(h_const.dim[2], THREAD_Z));
	gpu_hypterm_xy_stencil_kernel<<<grid_dim_xy_stencil, block_dim_xy_stencil>>>(d_const, d_cons, d_q, d_flux);

}


