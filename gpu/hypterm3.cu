#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "header.h"
#include "util.cuh"
#include "util.h"

#define BLOCK_SMALL		8
#define	BLOCK_LARGE		16

#define	s_q(i)			s_q[threadIdx.z+g->ng+(i)][threadIdx.x]
#define	s_qpres(i)		s_qpres[threadIdx.z+g->ng+(i)][threadIdx.x]
#define	s_cons(i, comp)	s_cons[comp][threadIdx.z+g->ng+(i)][threadIdx.x]

__global__ void gpu_hypterm_z_stencil_kernel(
	global_const_t *g,	// i:
	double *cons,		// i:
	double *q,			// i:
	double *flux		// o:
){
	int idx,si,sj,sk,tidx,tidz;
	double unp1, unp2, unp3, unp4, unm1, unm2, unm3, unm4;
	double flux_irho, flux_imx, flux_imy, flux_imz, flux_iene;

	__shared__ double       s_q[BLOCK_SMALL+NG+NG][BLOCK_LARGE];
	__shared__ double   s_qpres[BLOCK_SMALL+NG+NG][BLOCK_LARGE];
	__shared__ double s_cons[4][BLOCK_SMALL+NG+NG][BLOCK_LARGE];

	// Load to shared mem
	si = blockIdx.x*blockDim.x+threadIdx.x;
	sj = blockIdx.y*blockDim.y+threadIdx.y;
	sk = blockIdx.z*blockDim.z+threadIdx.z;

	tidx = threadIdx.x;
	tidz = threadIdx.z;
	while( tidz < BLOCK_SMALL+NG+NG && si < g->dim_g[0] && sj < g->dim_g[1] && sk < g->dim_g[2]){

		idx = sk*g->plane_offset_g_padded + (sj+g->ng)*g->pitch_g[0] + (si+g->ng);

				   s_q[tidz][tidx]  =     q[idx + qw*g->comp_offset_g_padded];
			   s_qpres[tidz][tidx]	=     q[idx + qpres*g->comp_offset_g_padded];
		 s_cons[s_imx][tidz][tidx] 	=  cons[idx + imx*g->comp_offset_g_padded];
		 s_cons[s_imy][tidz][tidx] 	=  cons[idx + imy*g->comp_offset_g_padded];
		 s_cons[s_imz][tidz][tidx] 	=  cons[idx + imz*g->comp_offset_g_padded];
		s_cons[s_iene][tidz][tidx] 	=  cons[idx + iene*g->comp_offset_g_padded];

		tidz += blockDim.z;
		sk   += blockDim.z;
	}
	__syncthreads();

	sk = blockIdx.z*blockDim.z+threadIdx.z;
	if(si < g->dim[0] && sj < g->dim[1] && sk < g->dim[2]){

		unp1 = s_q(1);	//q(i,j,k+1,qw);
		unp2 = s_q(2);	//q(i,j,k+2,qw);
		unp3 = s_q(3);	//q(i,j,k+3,qw);
		unp4 = s_q(4);	//q(i,j,k+4,qw);

		unm1 = s_q(-1);	//q(i,j,k-1,qw);
		unm2 = s_q(-2);	//q(i,j,k-2,qw);
		unm3 = s_q(-3);	//q(i,j,k-3,qw);
		unm4 = s_q(-4);	//q(i,j,k-4,qw);

		flux_irho =  ( g->ALP*(s_cons(1,s_imz)-s_cons(-1,s_imz))
					  + g->BET*(s_cons(2,s_imz)-s_cons(-2,s_imz))
					  + g->GAM*(s_cons(3,s_imz)-s_cons(-3,s_imz))
					  + g->DEL*(s_cons(4,s_imz)-s_cons(-4,s_imz)))*g->dxinv[2];

		flux_imx =   ( g->ALP*(s_cons(1,s_imx)*unp1-s_cons(-1,s_imx)*unm1)
					  + g->BET*(s_cons(2,s_imx)*unp2-s_cons(-2,s_imx)*unm2)
					  + g->GAM*(s_cons(3,s_imx)*unp3-s_cons(-3,s_imx)*unm3)
					  + g->DEL*(s_cons(4,s_imx)*unp4-s_cons(-4,s_imx)*unm4))*g->dxinv[2];

		flux_imy =   ( g->ALP*(s_cons(1,s_imy)*unp1-s_cons(-1,s_imy)*unm1)
					  + g->BET*(s_cons(2,s_imy)*unp2-s_cons(-2,s_imy)*unm2)
					  + g->GAM*(s_cons(3,s_imy)*unp3-s_cons(-3,s_imy)*unm3)
					  + g->DEL*(s_cons(4,s_imy)*unp4-s_cons(-4,s_imy)*unm4))*g->dxinv[2];

		flux_imz =   ( g->ALP*(s_cons(1,s_imz)*unp1-s_cons(-1,s_imz)*unm1
					  + (s_qpres(1)-s_qpres(-1)))
					  + g->BET*(s_cons(2,s_imz)*unp2-s_cons(-2,s_imz)*unm2
					  + (s_qpres(2)-s_qpres(-2)))
					  + g->GAM*(s_cons(3,s_imz)*unp3-s_cons(-3,s_imz)*unm3
					  + (s_qpres(3)-s_qpres(-3)))
					  + g->DEL*(s_cons(4,s_imz)*unp4-s_cons(-4,s_imz)*unm4
					  + (s_qpres(4)-s_qpres(-4))))*g->dxinv[2];

		flux_iene   = ( g->ALP*(s_cons(1,s_iene)*unp1-s_cons(-1,s_iene)*unm1
					  + (s_qpres(1)*unp1-s_qpres(-1)*unm1))
					  + g->BET*(s_cons(2,s_iene)*unp2-s_cons(-2,s_iene)*unm2
					  + (s_qpres(2)*unp2-s_qpres(-2)*unm2))
					  + g->GAM*(s_cons(3,s_iene)*unp3-s_cons(-3,s_iene)*unm3
					  + (s_qpres(3)*unp3-s_qpres(-3)*unm3))
					  + g->DEL*(s_cons(4,s_iene)*unp4-s_cons(-4,s_iene)*unm4
					  + (s_qpres(4)*unp4-s_qpres(-4)*unm4)))*g->dxinv[2];

		// Update changes
		idx = sk*g->plane_offset_padded + sj*g->pitch[0] + si;

		flux[idx + irho*g->comp_offset_padded] -= flux_irho;
		flux[idx + imx *g->comp_offset_padded] -= flux_imx;
		flux[idx + imy *g->comp_offset_padded] -= flux_imy;
		flux[idx + imz *g->comp_offset_padded] -= flux_imz;
		flux[idx + iene*g->comp_offset_padded] -= flux_iene;
	}
}
#undef	s_q
#undef 	s_qpres
#undef	s_cons


__global__ void gpu_hypterm_xy_stencil_kernel(
	global_const_t *g,	// i:
	double *cons,		// i:
	double *q,			// i:
	double *flux		// o:
){
	bool compute=false;
	int idx,out,si,sj,sk,tidx,tidy;
	double unp1, unp2, unp3, unp4, unm1, unm2, unm3, unm4;
	double flux_irho, flux_imx, flux_imy, flux_imz, flux_iene;

	__shared__ double      s_qu[BLOCK_LARGE+NG+NG][BLOCK_LARGE+NG+NG];
	__shared__ double      s_qv[BLOCK_LARGE+NG+NG][BLOCK_LARGE+NG+NG];
	__shared__ double   s_qpres[BLOCK_LARGE+NG+NG][BLOCK_LARGE+NG+NG];
	__shared__ double    s_cons[BLOCK_LARGE+NG+NG][BLOCK_LARGE+NG+NG];

	// Load to shared mem
	si = blockIdx.x*blockDim.x+threadIdx.x;
	sj = blockIdx.y*blockDim.y+threadIdx.y;
	sk = blockIdx.z*blockDim.z+threadIdx.z;

	out = sk*g->plane_offset_padded + sj*g->pitch[0] + si;
	compute = (si < g->dim[0] && sj < g->dim[1] && sk < g->dim[2]);

	for(sj=blockIdx.y*blockDim.y+threadIdx.y, tidy=threadIdx.y; tidy < BLOCK_LARGE+NG+NG; sj+=blockDim.y, tidy+=blockDim.y){
		for(si=blockIdx.x*blockDim.x+threadIdx.x, tidx=threadIdx.x; tidx < BLOCK_LARGE+NG+NG; si+=blockDim.x, tidx+=blockDim.x){
			if( si < g->dim_g[0] && sj < g->dim_g[1] && sk < g->dim_g[2]){

				idx = (sk+g->ng)*g->plane_offset_g_padded + sj*g->pitch_g[0] + si;

                          s_qu[tidy][tidx]  =     q[idx + qu*g->comp_offset_g_padded];
                          s_qv[tidy][tidx]  =     q[idx + qv*g->comp_offset_g_padded];
					   s_qpres[tidy][tidx]	=     q[idx + qpres*g->comp_offset_g_padded];
                        s_cons[tidy][tidx] 	=  cons[idx + imx*g->comp_offset_g_padded];
			}
		}
	}
	__syncthreads();

	if(compute){

#define	s_qu(i)			s_qu[threadIdx.y+g->ng][threadIdx.x+g->ng+(i)]
#define	s_qv(i)			s_qv[threadIdx.y+g->ng+(i)][threadIdx.x+g->ng]
#define	s_qpres_x(i)	s_qpres[threadIdx.y+g->ng][threadIdx.x+g->ng+(i)]
#define	s_qpres_y(i)	s_qpres[threadIdx.y+g->ng+(i)][threadIdx.x+g->ng]
#define	s_imx_x(i)	    s_cons[threadIdx.y+g->ng][threadIdx.x+g->ng+(i)]
#define	s_imx_y(i)	    s_cons[threadIdx.y+g->ng+(i)][threadIdx.x+g->ng]

        flux_irho = - ( g->ALP*(s_imx_x(1)-s_imx_x(-1))
                      + g->BET*(s_imx_x(2)-s_imx_x(-2))
                      + g->GAM*(s_imx_x(3)-s_imx_x(-3))
                      + g->DEL*(s_imx_x(4)-s_imx_x(-4)))*g->dxinv[0];

		flux_imx  = - ( g->ALP*(s_imx_x(1)*s_qu(1)-s_imx_x(-1)*s_qu(-1)
					  + (s_qpres_x(1)-s_qpres_x(-1)))
					  + g->BET*(s_imx_x(2)*s_qu(2)-s_imx_x(-2)*s_qu(-2)
					  + (s_qpres_x(2)-s_qpres_x(-2)))
					  + g->GAM*(s_imx_x(3)*s_qu(3)-s_imx_x(-3)*s_qu(-3)
					  + (s_qpres_x(3)-s_qpres_x(-3)))
					  + g->DEL*(s_imx_x(4)*s_qu(4)-s_imx_x(-4)*s_qu(-4)
					  + (s_qpres_x(4)-s_qpres_x(-4))))*g->dxinv[0];

		flux_imx -=   ( g->ALP*(s_imx_y(1)*s_qv(1)-s_imx_y(-1)*s_qv(-1))
					  + g->BET*(s_imx_y(2)*s_qv(2)-s_imx_y(-2)*s_qv(-2))
					  + g->GAM*(s_imx_y(3)*s_qv(3)-s_imx_y(-3)*s_qv(-3))
					  + g->DEL*(s_imx_y(4)*s_qv(4)-s_imx_y(-4)*s_qv(-4)))*g->dxinv[1];

        // Update changes
//		flux[out + imx *g->comp_offset_padded] = flux_imx;
	}
#undef s_imx_x
#undef s_imx_y
#define	s_imy_x(i)	    s_cons[threadIdx.y+g->ng][threadIdx.x+g->ng+(i)]
#define	s_imy_y(i)	    s_cons[threadIdx.y+g->ng+(i)][threadIdx.x+g->ng]

    __syncthreads();
	for(sj=blockIdx.y*blockDim.y+threadIdx.y, tidy=threadIdx.y; tidy < BLOCK_LARGE+NG+NG; sj+=blockDim.y, tidy+=blockDim.y){
		for(si=blockIdx.x*blockDim.x+threadIdx.x, tidx=threadIdx.x; tidx < BLOCK_LARGE+NG+NG; si+=blockDim.x, tidx+=blockDim.x){
			if( si < g->dim_g[0] && sj < g->dim_g[1] && sk < g->dim_g[2]){

				idx = (sk+g->ng)*g->plane_offset_g_padded + sj*g->pitch_g[0] + si;
                s_cons[tidy][tidx] 	=  cons[idx + imy*g->comp_offset_g_padded];
			}
		}
	}
	__syncthreads();

	if(compute){

        flux_irho -=  ( g->ALP*(s_imy_y(1)-s_imy_y(-1))
					  + g->BET*(s_imy_y(2)-s_imy_y(-2))
					  + g->GAM*(s_imy_y(3)-s_imy_y(-3))
					  + g->DEL*(s_imy_y(4)-s_imy_y(-4)))*g->dxinv[1];

        flux_imy =   -( g->ALP*(s_imy_y(1)*s_qv(1)-s_imy_y(-1)*s_qv(-1)
					  + (s_qpres_y(1)-s_qpres_y(-1)))
					  + g->BET*(s_imy_y(2)*s_qv(2)-s_imy_y(-2)*s_qv(-2)
					  + (s_qpres_y(2)-s_qpres_y(-2)))
					  + g->GAM*(s_imy_y(3)*s_qv(3)-s_imy_y(-3)*s_qv(-3)
					  + (s_qpres_y(3)-s_qpres_y(-3)))
					  + g->DEL*(s_imy_y(4)*s_qv(4)-s_imy_y(-4)*s_qv(-4)
					  + (s_qpres_y(4)-s_qpres_y(-4))))*g->dxinv[1];

		flux_imy  -=  ( g->ALP*(s_imy_x(1)*s_qu(1)-s_imy_x(-1)*s_qu(-1))
					  + g->BET*(s_imy_x(2)*s_qu(2)-s_imy_x(-2)*s_qu(-2))
					  + g->GAM*(s_imy_x(3)*s_qu(3)-s_imy_x(-3)*s_qu(-3))
					  + g->DEL*(s_imy_x(4)*s_qu(4)-s_imy_x(-4)*s_qu(-4)))*g->dxinv[0];

        // Update changes
//		flux[out + irho*g->comp_offset_padded] = flux_irho;
//		flux[out + imy*g->comp_offset_padded] = flux_imy;
	}
#undef  s_imy_x
#undef  s_imy_y
#define	s_imz_x(i)	    s_cons[threadIdx.y+g->ng][threadIdx.x+g->ng+(i)]
#define	s_imz_y(i)	    s_cons[threadIdx.y+g->ng+(i)][threadIdx.x+g->ng]

    __syncthreads();
	for(sj=blockIdx.y*blockDim.y+threadIdx.y, tidy=threadIdx.y; tidy < BLOCK_LARGE+NG+NG; sj+=blockDim.y, tidy+=blockDim.y){
		for(si=blockIdx.x*blockDim.x+threadIdx.x, tidx=threadIdx.x; tidx < BLOCK_LARGE+NG+NG; si+=blockDim.x, tidx+=blockDim.x){
			if( si < g->dim_g[0] && sj < g->dim_g[1] && sk < g->dim_g[2]){

				idx = (sk+g->ng)*g->plane_offset_g_padded + sj*g->pitch_g[0] + si;
                s_cons[tidy][tidx] 	=  cons[idx + imz*g->comp_offset_g_padded];
			}
		}
	}
	__syncthreads();

	if(compute){

		flux_imz  = - ( g->ALP*(s_imz_x(1)*s_qu(1)-s_imz_x(-1)*s_qu(-1))
					  + g->BET*(s_imz_x(2)*s_qu(2)-s_imz_x(-2)*s_qu(-2))
					  + g->GAM*(s_imz_x(3)*s_qu(3)-s_imz_x(-3)*s_qu(-3))
					  + g->DEL*(s_imz_x(4)*s_qu(4)-s_imz_x(-4)*s_qu(-4)))*g->dxinv[0];

		flux_imz -=   ( g->ALP*(s_imz_y(1)*s_qv(1)-s_imz_y(-1)*s_qv(-1))
					  + g->BET*(s_imz_y(2)*s_qv(2)-s_imz_y(-2)*s_qv(-2))
					  + g->GAM*(s_imz_y(3)*s_qv(3)-s_imz_y(-3)*s_qv(-3))
					  + g->DEL*(s_imz_y(4)*s_qv(4)-s_imz_y(-4)*s_qv(-4)))*g->dxinv[1];

        // Update changes
//		flux[out + imz*g->comp_offset_padded] = flux_imz;
	}

#undef  s_imz_x
#undef  s_imz_y
#define	s_iene_x(i)	    s_cons[threadIdx.y+g->ng][threadIdx.x+g->ng+(i)]
#define	s_iene_y(i)	    s_cons[threadIdx.y+g->ng+(i)][threadIdx.x+g->ng]

    __syncthreads();
	for(sj=blockIdx.y*blockDim.y+threadIdx.y, tidy=threadIdx.y; tidy < BLOCK_LARGE+NG+NG; sj+=blockDim.y, tidy+=blockDim.y){
		for(si=blockIdx.x*blockDim.x+threadIdx.x, tidx=threadIdx.x; tidx < BLOCK_LARGE+NG+NG; si+=blockDim.x, tidx+=blockDim.x){
			if( si < g->dim_g[0] && sj < g->dim_g[1] && sk < g->dim_g[2]){

				idx = (sk+g->ng)*g->plane_offset_g_padded + sj*g->pitch_g[0] + si;
                s_cons[tidy][tidx] 	=  cons[idx + iene*g->comp_offset_g_padded];
			}
		}
	}
	__syncthreads();

	if(compute){

        flux_iene =  -( g->ALP*(s_iene_x(1)*s_qu(1)-s_iene_x(-1)*s_qu(-1)
					  + (s_qpres_x(1)*s_qu(1)-s_qpres_x(-1)*s_qu(-1)))
					  + g->BET*(s_iene_x(2)*s_qu(2)-s_iene_x(-2)*s_qu(-2)
					  + (s_qpres_x(2)*s_qu(2)-s_qpres_x(-2)*s_qu(-2)))
					  + g->GAM*(s_iene_x(3)*s_qu(3)-s_iene_x(-3)*s_qu(-3)
					  + (s_qpres_x(3)*s_qu(3)-s_qpres_x(-3)*s_qu(-3)))
					  + g->DEL*(s_iene_x(4)*s_qu(4)-s_iene_x(-4)*s_qu(-4)
					  + (s_qpres_x(4)*s_qu(4)-s_qpres_x(-4)*s_qu(-4))))*g->dxinv[0];

        flux_iene -=  ( g->ALP*(s_iene_y(1)*s_qv(1)-s_iene_y(-1)*s_qv(-1)
					  + (s_qpres_y(1)*s_qv(1)-s_qpres_y(-1)*s_qv(-1)))
					  + g->BET*(s_iene_y(2)*s_qv(2)-s_iene_y(-2)*s_qv(-2)
					  + (s_qpres_y(2)*s_qv(2)-s_qpres_y(-2)*s_qv(-2)))
					  + g->GAM*(s_iene_y(3)*s_qv(3)-s_iene_y(-3)*s_qv(-3)
					  + (s_qpres_y(3)*s_qv(3)-s_qpres_y(-3)*s_qv(-3)))
					  + g->DEL*(s_iene_y(4)*s_qv(4)-s_iene_y(-4)*s_qv(-4)
					  + (s_qpres_y(4)*s_qv(4)-s_qpres_y(-4)*s_qv(-4))))*g->dxinv[1];



        // Update changes
//		flux[out + iene*g->comp_offset_padded] = flux_iene;
	}

#undef  s_iene_x
#undef  s_iene_y
#undef	s_qu
#undef  s_qv
#undef 	s_qpres_x
#undef 	s_qpres_y


	/** Z dimension **/
	si = blockIdx.x*blockDim.x+threadIdx.x;
	sj = blockIdx.y*blockDim.y+threadIdx.y;
	if(compute){

		idx = (sk+g->ng)*g->plane_offset_g_padded + (sj+g->ng)*g->pitch_g[0] + si+g->ng;

		double imxp1, imxp2, imxp3, imxp4;
		double imxm1, imxm2, imxm3, imxm4;
		double imyp1, imyp2, imyp3, imyp4;
		double imym1, imym2, imym3, imym4;
		double imzp1, imzp2, imzp3, imzp4;
		double imzm1, imzm2, imzm3, imzm4;
		double ienep1, ienep2, ienep3, ienep4;
		double ienem1, ienem2, ienem3, ienem4;
		double qpresp1, qpresp2, qpresp3, qpresp4;
		double qpresm1, qpresm2, qpresm3, qpresm4;

		unp1 = q[qw*g->comp_offset_g_padded + idx + 1*g->plane_offset_g_padded];
		unp2 = q[qw*g->comp_offset_g_padded + idx + 2*g->plane_offset_g_padded];
		unp3 = q[qw*g->comp_offset_g_padded + idx + 3*g->plane_offset_g_padded];
		unp4 = q[qw*g->comp_offset_g_padded + idx + 4*g->plane_offset_g_padded];
		unm1 = q[qw*g->comp_offset_g_padded + idx - 1*g->plane_offset_g_padded];
		unm2 = q[qw*g->comp_offset_g_padded + idx - 2*g->plane_offset_g_padded];
		unm3 = q[qw*g->comp_offset_g_padded + idx - 3*g->plane_offset_g_padded];
		unm4 = q[qw*g->comp_offset_g_padded + idx - 4*g->plane_offset_g_padded];

		imxp1 = cons[imx*g->comp_offset_g_padded + idx + 1*g->plane_offset_g_padded];
		imxp2 = cons[imx*g->comp_offset_g_padded + idx + 2*g->plane_offset_g_padded];
		imxp3 = cons[imx*g->comp_offset_g_padded + idx + 3*g->plane_offset_g_padded];
		imxp4 = cons[imx*g->comp_offset_g_padded + idx + 4*g->plane_offset_g_padded];
		imxm1 = cons[imx*g->comp_offset_g_padded + idx - 1*g->plane_offset_g_padded];
		imxm2 = cons[imx*g->comp_offset_g_padded + idx - 2*g->plane_offset_g_padded];
		imxm3 = cons[imx*g->comp_offset_g_padded + idx - 3*g->plane_offset_g_padded];
		imxm4 = cons[imx*g->comp_offset_g_padded + idx - 4*g->plane_offset_g_padded];

		imyp1 = cons[imy*g->comp_offset_g_padded + idx + 1*g->plane_offset_g_padded];
		imyp2 = cons[imy*g->comp_offset_g_padded + idx + 2*g->plane_offset_g_padded];
		imyp3 = cons[imy*g->comp_offset_g_padded + idx + 3*g->plane_offset_g_padded];
		imyp4 = cons[imy*g->comp_offset_g_padded + idx + 4*g->plane_offset_g_padded];
		imym1 = cons[imy*g->comp_offset_g_padded + idx - 1*g->plane_offset_g_padded];
		imym2 = cons[imy*g->comp_offset_g_padded + idx - 2*g->plane_offset_g_padded];
		imym3 = cons[imy*g->comp_offset_g_padded + idx - 3*g->plane_offset_g_padded];
		imym4 = cons[imy*g->comp_offset_g_padded + idx - 4*g->plane_offset_g_padded];

		imzp1 = cons[imz*g->comp_offset_g_padded + idx + 1*g->plane_offset_g_padded];
		imzp2 = cons[imz*g->comp_offset_g_padded + idx + 2*g->plane_offset_g_padded];
		imzp3 = cons[imz*g->comp_offset_g_padded + idx + 3*g->plane_offset_g_padded];
		imzp4 = cons[imz*g->comp_offset_g_padded + idx + 4*g->plane_offset_g_padded];
		imzm1 = cons[imz*g->comp_offset_g_padded + idx - 1*g->plane_offset_g_padded];
		imzm2 = cons[imz*g->comp_offset_g_padded + idx - 2*g->plane_offset_g_padded];
		imzm3 = cons[imz*g->comp_offset_g_padded + idx - 3*g->plane_offset_g_padded];
		imzm4 = cons[imz*g->comp_offset_g_padded + idx - 4*g->plane_offset_g_padded];

		ienep1 = cons[iene*g->comp_offset_g_padded + idx + 1*g->plane_offset_g_padded];
		ienep2 = cons[iene*g->comp_offset_g_padded + idx + 2*g->plane_offset_g_padded];
		ienep3 = cons[iene*g->comp_offset_g_padded + idx + 3*g->plane_offset_g_padded];
		ienep4 = cons[iene*g->comp_offset_g_padded + idx + 4*g->plane_offset_g_padded];
		ienem1 = cons[iene*g->comp_offset_g_padded + idx - 1*g->plane_offset_g_padded];
		ienem2 = cons[iene*g->comp_offset_g_padded + idx - 2*g->plane_offset_g_padded];
		ienem3 = cons[iene*g->comp_offset_g_padded + idx - 3*g->plane_offset_g_padded];
		ienem4 = cons[iene*g->comp_offset_g_padded + idx - 4*g->plane_offset_g_padded];

		qpresp1 = q[qpres*g->comp_offset_g_padded + idx + 1*g->plane_offset_g_padded];
		qpresp2 = q[qpres*g->comp_offset_g_padded + idx + 2*g->plane_offset_g_padded];
		qpresp3 = q[qpres*g->comp_offset_g_padded + idx + 3*g->plane_offset_g_padded];
		qpresp4 = q[qpres*g->comp_offset_g_padded + idx + 4*g->plane_offset_g_padded];
		qpresm1 = q[qpres*g->comp_offset_g_padded + idx - 1*g->plane_offset_g_padded];
		qpresm2 = q[qpres*g->comp_offset_g_padded + idx - 2*g->plane_offset_g_padded];
		qpresm3 = q[qpres*g->comp_offset_g_padded + idx - 3*g->plane_offset_g_padded];
		qpresm4 = q[qpres*g->comp_offset_g_padded + idx - 4*g->plane_offset_g_padded];


		flux_irho -=  ( g->ALP*(imzp1-imzm1)
					  + g->BET*(imzp2-imzm2)
					  + g->GAM*(imzp3-imzm3)
					  + g->DEL*(imzp4-imzm4))*g->dxinv[2];

		flux_imx -=   ( g->ALP*(imxp1*unp1-imxm1*unm1)
					  + g->BET*(imxp2*unp2-imxm2*unm2)
					  + g->GAM*(imxp3*unp3-imxm3*unm3)
					  + g->DEL*(imxp4*unp4-imxm4*unm4))*g->dxinv[2];

		flux_imy -=   ( g->ALP*(imyp1*unp1-imym1*unm1)
					  + g->BET*(imyp2*unp2-imym2*unm2)
					  + g->GAM*(imyp3*unp3-imym3*unm3)
					  + g->DEL*(imyp4*unp4-imym4*unm4))*g->dxinv[2];

		flux_imz -=   ( g->ALP*(imzp1*unp1-imzm1*unm1
					  + (qpresp1-qpresm1))
					  + g->BET*(imzp2*unp2-imzm2*unm2
					  + (qpresp2-qpresm2))
					  + g->GAM*(imzp3*unp3-imzm3*unm3
					  + (qpresp3-qpresm3))
					  + g->DEL*(imzp4*unp4-imzm4*unm4
					  + (qpresp4-qpresm4)))*g->dxinv[2];

		flux_iene -=  ( g->ALP*(ienep1*unp1-ienem1*unm1
					  + (qpresp1*unp1-qpresm1*unm1))
					  + g->BET*(ienep2*unp2-ienem2*unm2
					  + (qpresp2*unp2-qpresm2*unm2))
					  + g->GAM*(ienep3*unp3-ienem3*unm3
					  + (qpresp3*unp3-qpresm3*unm3))
					  + g->DEL*(ienep4*unp4-ienem4*unm4
					  + (qpresp4*unp4-qpresm4*unm4)))*g->dxinv[2];

		// Update global memory
		flux[out + irho*g->comp_offset_padded] = flux_irho;
		flux[out + imx *g->comp_offset_padded] = flux_imx;
		flux[out + imy*g->comp_offset_padded] = flux_imy;
		flux[out + imz*g->comp_offset_padded] = flux_imz;
		flux[out + iene*g->comp_offset_padded] = flux_iene;
	}
}

void gpu_hypterm3(
	global_const_t h_const, 	// i: Global struct containing application parameters
	global_const_t *d_const,	// i: Device pointer to global struct containing application paramters
	double *d_cons,				// i:
	double *d_q,				// i:
	double *d_flux				// o: flux
){

	/** d_flux must be set to zero beforehand (in diffterm, etc) **/

	dim3 block_dim_xy_stencil(BLOCK_LARGE, BLOCK_LARGE, 1);
	dim3 grid_dim_xy_stencil(CEIL(h_const.dim[0], BLOCK_LARGE), CEIL(h_const.dim[1], BLOCK_LARGE), h_const.dim[2]);
	gpu_hypterm_xy_stencil_kernel<<<grid_dim_xy_stencil, block_dim_xy_stencil>>>(d_const, d_cons, d_q, d_flux);

}


